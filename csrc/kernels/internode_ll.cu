#include "hip/hip_runtime.h"
#include "configs.cuh"
#include "exception.cuh"
#include "launch.cuh"
#include "buffer.cuh"
#include "utils.cuh"
#include "shmem_wrapper.cuh"
#include <hip/hip_cooperative_groups.h>
#include <rocshmem/rocshmem.hpp>
#include <iostream>
// low latency+RocSHMEM has issue with CTX.
#define ROCM_DISABLE_CTX

namespace cg = cooperative_groups;
using namespace rocshmem;
namespace deep_ep {

namespace internode_ll {

__device__ void grid_barrier(int* global_counter, int num_blocks) {
volatile int ret;
    __syncthreads();
    __threadfence();
    if (threadIdx.x == 0 ) {
        ret = __hip_atomic_fetch_add( &global_counter[0], 1,
                            __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_AGENT);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
            while (__hip_atomic_load(global_counter,
                                     __ATOMIC_RELAXED,
                                     __HIP_MEMORY_SCOPE_AGENT) != num_blocks);
    }
    __syncthreads();
}


template <int kNumThreads> __launch_bounds__(kNumThreads, 1)
__global__ void clean_low_latency_buffer(int64_t* clean_0, int num_clean_int_0,
                                         int64_t* clean_1, int num_clean_int_1) {
    // Barrier before cleaning (in case of unfinished chunked EP)
#ifdef USE_ROCM
    if (threadIdx.x == 0)
        internode::shmem_device_barrier_all();
#else
    nvshmemx_barrier_all_block();
#endif

    // Clean
    auto thread_id = static_cast<int>(threadIdx.x);
    #pragma unroll
    for (int i = thread_id; i < num_clean_int_0; i += kNumThreads)
        clean_0[i] = 0;
    #pragma unroll
    for (int i = thread_id; i < num_clean_int_1; i += kNumThreads)
        clean_1[i] = 0;

    // Barrier after cleaning (make sure low-latency mode work 
#ifdef USE_ROCM
    if (threadIdx.x == 0)
        internode::shmem_device_barrier_all();
#else
    nvshmemx_barrier_all_block();
#endif
}

void clean_low_latency_buffer(int64_t* clean_0, int num_clean_int_0,
                              int64_t* clean_1, int num_clean_int_1,
                              hipStream_t stream) {
    constexpr int kNumThreads = 256;

    SETUP_LAUNCH_CONFIG(1, kNumThreads, stream);
    LAUNCH_KERNEL_NON_COOPERATIVE(&cfg, clean_low_latency_buffer<kNumThreads>,
                  clean_0, num_clean_int_0, clean_1, num_clean_int_1);
}

template <bool kUseFP8, int kNumWarpGroups, int kNumWarpsPerGroup, int kHidden>
__global__ __launch_bounds__(kNumWarpGroups * kNumWarpsPerGroup * kWarpSize, 1) void
dispatch(void* packed_recv_x, float* packed_recv_x_scales,
         int* packed_recv_src_info, int64_t* packed_recv_layout_range,
         int* packed_recv_count,
         int* global_atomic_counter,
         void* rdma_recv_x, int64_t* rdma_recv_count, void* rdma_x,
         const void* x, const int64_t* topk_idx,
         int* atomic_counter_per_expert, int* atomic_finish_counter_per_expert,
         int64_t* next_clean, int num_next_clean_int,
         int num_tokens, int num_max_dispatch_tokens_per_rank,
         int num_topk, int num_experts, int rank, int num_ranks,
         int phases) {
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto warp_id = thread_id / kWarpSize, lane_id = get_lane_id();
    const auto num_sms = static_cast<int>(gridDim.x);
    const auto num_warps = kNumWarpGroups * kNumWarpsPerGroup;
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / kNumWarpsPerGroup;
    const auto sub_warp_id = warp_id % kNumWarpsPerGroup;
    const auto responsible_expert_idx = sm_id * kNumWarpGroups + warp_group_id;

#if !defined(ROCM_DISABLE_CTX)
    __shared__ internode::shmem_ctx_t ctx;
    internode::shmem_wg_ctx_create(&ctx);
#endif

    // FP8 staffs
    constexpr int kNumPerChannels = 128;
#ifdef USE_ROCM
    constexpr float kFP8Margin = 1e-4, kFP8Amax = 240, kFP8AmaxInv = 1.0f / 240.0f;
    const size_t hidden_bytes = kHidden * (kUseFP8 ? sizeof(__hip_fp8_storage_t) : sizeof(gpu_bfloat16_t));
#else
    constexpr float kFP8Margin = 1e-4, kFP8Amax = 448, kFP8AmaxInv = 1.0f / 448.0f;
    const size_t hidden_bytes = kHidden * (kUseFP8 ? sizeof(__hip_fp8_storage_t) : sizeof(gpu_bfloat16_t));
#endif
    const int num_scales = kHidden / kNumPerChannels;
    const size_t hidden_int4 = hidden_bytes / sizeof(int4);

    // Message package: hidden data, FP8 scales, index at source
    // NOTES: currently we have 3 reserved int fields for future use
    using vec_t = typename std::conditional<kUseFP8, int2, int4>::type;
    const size_t num_bytes_per_msg = sizeof(int4) + (kUseFP8 ? (kHidden + num_scales * sizeof(float)) : (kHidden * sizeof(gpu_bfloat16_t)));
    const size_t num_int4_per_msg = num_bytes_per_msg / sizeof(int4);
    EP_DEVICE_ASSERT(num_bytes_per_msg % sizeof(int4) == 0);

#ifdef USE_ROCM
    // 16 is the max possible number of warps in AMD GPUs 
    constexpr int kMaxNumWarps = 1024 / kWarpSize;
    constexpr int num_sync_large_iteration = kMaxNumWarps ;
    __shared__ volatile uint8_t sync_large_warp_counters[num_sync_large_iteration];

    #pragma unroll
    for (int i = thread_id; i < num_sync_large_iteration; i += blockDim.x) {
        sync_large_warp_counters[i] = 0;
    }
    __syncthreads();
#endif
    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_DISPATCH_RECV;

    // Expert counts
    __shared__ int shared_num_tokens_sent_per_expert[kNumWarpGroups];

    // There are 2 kinds of warps in this part:
    // 1. The first-kind warps for FP8 cast and sending top-k tokens
    // 2. The last warp for reading `topk_idx` and count for per-expert information
    if (warp_id < num_warps ) {
        constexpr int kNumElemsPerRead = sizeof(int4) / sizeof(gpu_bfloat16_t);
        EP_DEVICE_ASSERT(kHidden % kNumElemsPerRead == 0);
        EP_STATIC_ASSERT(kNumElemsPerRead * kWarpSize % kNumPerChannels == 0, "Invalid vectorization");
        const auto num_threads = (num_warps - 1) * kWarpSize;
        const size_t hidden_bf16_int4 = kHidden / kNumElemsPerRead;

        for (int token_idx = sm_id; token_idx < num_tokens; token_idx += num_sms) {
            const auto x_int4 = reinterpret_cast<const int4*>(x) + token_idx * hidden_bf16_int4;
            const auto rdma_x_src_idx = reinterpret_cast<int*>(reinterpret_cast<uint8_t*>(rdma_x) + token_idx * num_bytes_per_msg);
            const auto rdma_x_vec = reinterpret_cast<vec_t*>(reinterpret_cast<uint8_t*>(rdma_x_src_idx) + sizeof(int4));
            const auto rdma_x_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(rdma_x_vec) + hidden_bytes);

            // Overlap top-k index read and source token index write
            auto dst_expert_idx = warp_id < num_topk ? static_cast<int>(__ldg(topk_idx + token_idx * num_topk + warp_id)) : -1;
            thread_id == 0 ? (*rdma_x_src_idx = token_idx) : 0;

            // FP8 cast
            #pragma unroll
            for (int i = thread_id; i < hidden_bf16_int4; i += num_threads) {
                // Read
                auto int4_value = __ldg(x_int4 + i);

                if (kUseFP8) {
                    // Calculate local amax
                    auto bf16_values = reinterpret_cast<gpu_bfloat16_t*>(&int4_value);
                    float fp32_values[kNumElemsPerRead];
                    float amax = kFP8Margin, scale, scale_inv;
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; ++ j) {
                        fp32_values[j] = static_cast<float>(bf16_values[j]);
                        amax = fmaxf(amax, fabsf(fp32_values[j]));
                    }
#ifdef USE_ROCM
                    // Reduce amax and scale
                    EP_STATIC_ASSERT(kNumElemsPerRead * kWarpSize / kNumPerChannels == 4, "Invalid vectorization");
                    amax = quarter_warp_reduce_max(amax), scale = kFP8Amax / amax, scale_inv = amax * kFP8AmaxInv;
                    if (lane_id % 16 == 0)
#else
                    EP_STATIC_ASSERT(kNumElemsPerRead * kWarpSize / kNumPerChannels == 2, "Invalid vectorization");
                    amax = half_warp_reduce_max(amax), scale = kFP8Amax / amax, scale_inv = amax * kFP8AmaxInv;
                    if (lane_id == 0 or lane_id == 16)
#endif
                        rdma_x_scales[i * kNumElemsPerRead / 128] = scale_inv;

                    // Cast into send buffer
                    vec_t int2_value;
#ifdef USE_ROCM
                    auto fp8x2_values = reinterpret_cast<__hip_fp8x2_storage_t*>(&int2_value);
#else
                    auto fp8x2_values = reinterpret_cast<__hip_fp8x2_storage_t*>(&int2_value);
#endif
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; j += 2) {
                        float2 fp32x2 = {fp32_values[j] * scale, fp32_values[j + 1] * scale};
#ifdef USE_ROCM
#if defined(__gfx942__)
                        fp8x2_values[j / 2] = __hip_cvt_float2_to_fp8x2(fp32x2, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
#endif
#if defined(__gfx950__)
                        fp8x2_values[j / 2] = __hip_cvt_float2_to_fp8x2(fp32x2, __HIP_SATFINITE, __HIP_E4M3);
#endif
#else
                        fp8x2_values[j / 2] = __hip_cvt_float2_to_fp8x2(fp32x2, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
#endif
                    }
                    rdma_x_vec[i] = int2_value;
                } else {
                    // Reinterpret-cast is for C++14 compatibility
                    rdma_x_vec[i] = *reinterpret_cast<vec_t*>(&int4_value);
                }
            }
#ifdef USE_ROCM
            __syncthreads();
#else
            asm volatile("bar.sync 1, %0;" :: "r"(num_threads));
#endif
            // Issue IBGDA sends
            if (dst_expert_idx >= 0) {
                int slot_idx = lane_id == 0 ? atomicAdd(atomic_counter_per_expert + dst_expert_idx, 1) : 0;
                slot_idx = shfl_sync(slot_idx, 0);
                const auto dst_rank = dst_expert_idx / num_local_experts;
                const auto dst_expert_local_idx = dst_expert_idx % num_local_experts;
                const auto src_ptr = reinterpret_cast<uint64_t>(rdma_x_src_idx);
                const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) +
                                     dst_expert_local_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                                     rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                                     slot_idx * num_bytes_per_msg;
                if (dst_rank != rank) {

#ifdef USE_ROCM
#if defined(ROCM_DISABLE_CTX)
                    internode::shmemx_int8_put_nbi_warp(
#else
                    internode::shmem_ctx_schar_put_nbi_warp(ctx,
#endif
                    reinterpret_cast<signed char*>(dst_ptr), reinterpret_cast<signed char*>(src_ptr), num_bytes_per_msg, dst_rank);
#else
                    nvshmemi_ibgda_put_nbi_warp(dst_ptr, src_ptr, num_bytes_per_msg, dst_rank, dst_expert_local_idx, lane_id, slot_idx);
#endif
                } else {
                    // NOTES: only 2 load iterations for 7K hidden with 8 unrolls
                    const auto* src_int4_ptr = reinterpret_cast<const int4*>(src_ptr);
                    const auto* dst_int4_ptr = reinterpret_cast<int4*>(dst_ptr);
                    UNROLLED_WARP_COPY(8, lane_id, num_int4_per_msg, dst_int4_ptr, src_int4_ptr, ld_nc_global, st_na_global);
                }

                // Increase counter after finishing
                syncwarp();
                lane_id == 0 ? atomic_add_release_global(atomic_finish_counter_per_expert + dst_expert_idx, 1) : 0;
            }
        }
    } if (warp_id == num_warps - 1) {
        EP_DEVICE_ASSERT(num_sms > 1);
        if (sm_id == 0) {
            // The first SM is also responsible for checking QPs
#ifndef USE_ROCM
            EP_DEVICE_ASSERT(ibgda_get_state()->num_rc_per_pe == num_local_experts);
#endif
            // The first SM is also responsible for cleaning the next buffer
            #pragma unroll
            for (int i = lane_id; i < num_next_clean_int; i += kWarpSize)
                next_clean[i] = 0;

            // Notify before executing `int_p`
            syncwarp();
            #pragma unroll
            for (int i = lane_id; i < num_experts; i += kWarpSize)
                atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG);
        }
        // This SM should be responsible for some destination experts, read `topk_idx` for them
        int expert_count[kNumWarpGroups] = {0};
        const auto expert_begin_idx = sm_id * kNumWarpGroups;
        const auto expert_end_idx = min(expert_begin_idx + kNumWarpGroups, num_experts);

        // Per lane count
        #pragma unroll 8
        for (int i = lane_id; i < num_tokens * num_topk; i += kWarpSize) {
            auto idx = static_cast<int>(__ldg(topk_idx + i));
            if (idx >= expert_begin_idx and idx < expert_end_idx)
                expert_count[idx - expert_begin_idx] ++;
        }

        // Warp reduce
        #pragma unroll
        for (int i = expert_begin_idx; i < expert_end_idx; ++ i) {
            auto sum = warp_reduce_sum(expert_count[i - expert_begin_idx]);
            if (lane_id == 0) {
                shared_num_tokens_sent_per_expert[i - expert_begin_idx] = sum;
                atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG - sum);
            }
        }
    }

    if (thread_id == 0 and num_ranks > 8){
#if defined(ROCM_DISABLE_CTX)
                    internode::shmem_fence();
#else
                    internode::shmem_ctx_quiet(ctx);
#endif
    }
    //revert sync_large_warp_counters to 0 for next sync
    __syncthreads();

    // Issue count sends
    if (responsible_expert_idx < num_experts and sub_warp_id == 0 and lane_id == 0) {
        const auto dst_rank = responsible_expert_idx / num_local_experts;
        const auto dst_expert_local_idx = responsible_expert_idx % num_local_experts;
        const auto num_tokens_sent = shared_num_tokens_sent_per_expert[responsible_expert_idx - sm_id * kNumWarpGroups];

        // Wait local sends issued and send expert counts
        while (ld_acquire_global(atomic_finish_counter_per_expert + responsible_expert_idx) != FINISHED_SUM_TAG * 2);
        if (dst_rank != rank) {
#ifdef USE_ROCM
#if defined(ROCM_DISABLE_CTX)
           internode::shmem_long_atomic_add( rdma_recv_count + dst_expert_local_idx * num_ranks + rank, -num_tokens_sent - 1, dst_rank);
#else
           internode::shmem_ctx_long_atomic_add(ctx, rdma_recv_count + dst_expert_local_idx * num_ranks + rank, -num_tokens_sent - 1, dst_rank);
#endif
#else //CUDA
           nvshmemi_ibgda_amo_nonfetch_add(rdma_recv_count + dst_expert_local_idx * num_ranks + rank, -num_tokens_sent - 1, dst_rank, dst_expert_local_idx);
#endif
        } else {
            st_na_release(reinterpret_cast<int64_t *>(rdma_recv_count + dst_expert_local_idx * num_ranks + rank), -num_tokens_sent - 1);
        }

        // Clean workspace for next use
        atomic_counter_per_expert[responsible_expert_idx] = 0;
        atomic_finish_counter_per_expert[responsible_expert_idx] = 0;

        // Clean `packed_recv_count`
        if (dst_rank == 0)
            packed_recv_count[dst_expert_local_idx] = 0;
    }
    syncwarp();

    // Receiving phase
    LOW_LATENCY_DISPATCH_RECV:
    if ((phases & LOW_LATENCY_RECV_PHASE) == 0)
        return;

    // For send-and-recv kernels, we need a grid sync for making `packed_recv_count` visible
    if (phases & LOW_LATENCY_SEND_PHASE){
        grid_barrier(global_atomic_counter, num_sms);
    }
    // Receiving and packing
    if (responsible_expert_idx < num_experts) {
        const auto src_rank = responsible_expert_idx / num_local_experts;
        const auto local_expert_idx = responsible_expert_idx % num_local_experts;
        const auto rdma_recv_x_uint8 = reinterpret_cast<uint8_t*>(rdma_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                src_rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg;
        const auto recv_x_int4 = reinterpret_cast<int4*>(packed_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_int4;
        const auto recv_x_scales = packed_recv_x_scales + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_scales;
        const auto recv_src_info = packed_recv_src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
        const auto recv_range = packed_recv_layout_range + local_expert_idx * num_ranks;

        // Shared between sub-warps in warp groups
        __shared__ int shared_num_recv_tokens[kNumWarpGroups], shared_recv_token_begin_idx[kNumWarpGroups];

        // Wait tokens to arrive
        // NOTES: using sub-warp 1 to overlap with sub-warp 0
        int num_recv_tokens, recv_token_begin_idx;
        EP_STATIC_ASSERT(kNumWarpsPerGroup > 1, "Requires more than one warp per group");
        if (sub_warp_id == 1 and lane_id == 0) {
            while ((num_recv_tokens = ld_acquire_global(reinterpret_cast<int*>(rdma_recv_count + local_expert_idx * num_ranks + src_rank))) == 0);
            num_recv_tokens = -num_recv_tokens - 1;
            recv_token_begin_idx = atomicAdd(packed_recv_count + local_expert_idx, num_recv_tokens);
            shared_num_recv_tokens[warp_group_id] = num_recv_tokens;
            shared_recv_token_begin_idx[warp_group_id] = recv_token_begin_idx;
            recv_range[src_rank] = pack2<int, int64_t>(num_recv_tokens, recv_token_begin_idx);
        }
#ifdef USE_ROCM
        // no needs to reset because there is no iteration
        if (lane_id == 0){
            volatile int ret = __hip_atomic_fetch_add(
                &sync_large_warp_counters[warp_group_id], 1,
                __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
        }
        syncwarp();
        while (sync_large_warp_counters[warp_group_id] < (kNumWarpsPerGroup));
#else
        asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 2), "r"(kNumWarpsPerGroup * 32));
#endif
        num_recv_tokens = shared_num_recv_tokens[warp_group_id];
        recv_token_begin_idx = shared_recv_token_begin_idx[warp_group_id];

        // Copy tokens
        EP_DEVICE_ASSERT(num_scales <= 64);
        for (int i = sub_warp_id; i < num_recv_tokens; i += kNumWarpsPerGroup) {
            // Copy source info
            const auto src_src_idx = reinterpret_cast<int*>(rdma_recv_x_uint8 + i * num_bytes_per_msg);
            if (lane_id == 0)
                recv_src_info[recv_token_begin_idx + i] = ld_nc_global(src_src_idx);
            syncwarp();

            // Copy data
            // NOTES: only 2 load iterations for 7K hidden with 7 unrolls
            const auto src_data = reinterpret_cast<int4*>(reinterpret_cast<uint8_t*>(src_src_idx) + sizeof(int4));
            const auto dst_data = recv_x_int4 + (recv_token_begin_idx + i) * hidden_int4;
            UNROLLED_WARP_COPY(7, lane_id, hidden_int4, dst_data, src_data, ld_nc_global, st_na_global);

            // Copy scales
            if (kUseFP8) {
                const auto src_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(src_data) + hidden_bytes);
                const auto dst_scales = reinterpret_cast<float*>(recv_x_scales + recv_token_begin_idx + i);
                const auto scale_stride = num_ranks * num_max_dispatch_tokens_per_rank;
                auto scale_0 = lane_id < num_scales ? ld_nc_global(src_scales + lane_id) : 0;
                auto scale_1 = (lane_id + kWarpSize) < num_scales ? ld_nc_global(src_scales + lane_id + kWarpSize) : 0;
                lane_id < num_scales ? dst_scales[lane_id * scale_stride] = scale_0 : 0.0f;
                (lane_id + kWarpSize) < num_scales ? dst_scales[(lane_id + kWarpSize) * scale_stride] = scale_1 : 0.0f;
            }
        }
    }
#if !defined(ROCM_DISABLE_CTX)
    internode::shmem_wg_ctx_destroy(&ctx);
#endif
}

void dispatch(void* packed_recv_x, float* packed_recv_x_scales,
              int* packed_recv_src_info, int64_t* packed_recv_layout_range,
              int* packed_recv_count,
              int* global_atomic_counter,
              void* rdma_recv_x, int64_t* rdma_recv_count, void* rdma_x,
              const void* x, const int64_t* topk_idx,
              int64_t* next_clean, int num_next_clean_int,
              int num_tokens, int hidden, int num_max_dispatch_tokens_per_rank,
              int num_topk, int num_experts, int rank, int num_ranks, bool use_fp8,
              void* workspace, hipStream_t stream, int phases) {

#ifdef USE_ROCM
    constexpr int kNumWarpsPerGroup = 5;
    constexpr int kNumWarpGroups = 2;
#else
    constexpr int kNumWarpsPerGroup = 10;
    constexpr int kNumWarpGroups = 3;
#endif
    constexpr int kNumMaxTopK = 9;
    EP_STATIC_ASSERT(kNumMaxTopK + 1 <= kNumWarpGroups * kNumWarpsPerGroup, "Too many top-k selections");

    const auto num_warps = kNumWarpGroups * kNumWarpsPerGroup;
    const auto num_sms = cell_div(num_experts, kNumWarpGroups);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopK);

    // Workspace checks
    auto atomic_counter_per_expert = reinterpret_cast<int*>(workspace);
    auto atomic_finish_counter_per_expert = atomic_counter_per_expert + num_experts;
    EP_HOST_ASSERT(num_experts * sizeof(int) * 2 <= NUM_WORKSPACE_BYTES);

#define DISPATCH_LAUNCH_CASE(hidden) { \
auto dispatch_func = use_fp8 ? dispatch<true, kNumWarpGroups, kNumWarpsPerGroup, hidden> : \
                               dispatch<false, kNumWarpGroups, kNumWarpsPerGroup, hidden>; \
LAUNCH_KERNEL_NON_COOPERATIVE(&cfg, dispatch_func, \
              packed_recv_x, packed_recv_x_scales, \
              packed_recv_src_info, packed_recv_layout_range, \
              packed_recv_count, \
              global_atomic_counter, \
              rdma_recv_x, rdma_recv_count, rdma_x, \
              x, topk_idx, \
              atomic_counter_per_expert, atomic_finish_counter_per_expert, \
              next_clean, num_next_clean_int, \
              num_tokens, num_max_dispatch_tokens_per_rank, \
              num_topk, num_experts, rank, num_ranks, phases); } break

    SETUP_LAUNCH_CONFIG(num_sms, num_warps * kWarpSize, stream);
    SWITCH_HIDDEN(DISPATCH_LAUNCH_CASE);
#undef DISPATCH_LAUNCH_CASE
}

template <int kNumWarpGroups, int kNumWarpsPerGroup, int kHidden, int kNumMaxTopk>
__global__ __launch_bounds__(kNumWarpGroups * kNumWarpsPerGroup * kWarpSize, 1) void
combine(void* combined_x,
        void* rdma_recv_x, int64_t* rdma_recv_flag, void* rdma_send_x,
        const void* x, const int64_t* topk_idx, const float* topk_weights,
        const int* src_info, const int64_t* layout_range,
        int* global_atomic_counter,
        int64_t* next_clean, int num_next_clean_int,
        int* atomic_clean_flag,
        int num_combined_tokens, int hidden, int num_topk,
        int num_max_dispatch_tokens_per_rank,
        int num_experts, int rank, int num_ranks,
        int phases, bool zero_copy) {

#if !defined(ROCM_DISABLE_CTX)
    __shared__ internode::shmem_ctx_t ctx;
    internode::shmem_wg_ctx_create(&ctx);
#endif
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto num_sms = static_cast<int>(gridDim.x);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto num_threads = static_cast<int>(blockDim.x);
    const auto warp_id = thread_id / kWarpSize, lane_id = get_lane_id();
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / kNumWarpsPerGroup;
    const auto sub_warp_id = warp_id % kNumWarpsPerGroup;
    const auto responsible_expert_idx = sm_id * kNumWarpGroups + warp_group_id;

    // Data type staffs
    constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(gpu_bfloat16_t);
    const size_t hidden_bf16_int4 = kHidden / kNumElemsPerInt4;

    // Message package
    // BF16 mode: always use BF16 for hidden data (ignoring the extra flag slot)
    constexpr size_t num_bytes_per_slot = sizeof(int4) + kHidden * sizeof(gpu_bfloat16_t);
    EP_STATIC_ASSERT(num_bytes_per_slot % sizeof(int4) == 0, "Invalid vectorization");
    __syncthreads();
#ifdef USE_ROCM
    // 16 is the max possible number of warps in AMD GPUs 
    constexpr int kMaxNumWarps = 1024 / kWarpSize;
    __shared__ volatile int sync_large_warp_counters[kMaxNumWarps];
    if (threadIdx.x==0){
        // printf("combine");
        #pragma unroll
        for (int i = 0; i < kMaxNumWarps; ++i) {
            sync_large_warp_counters[i] = 0;
        }
    }
    __syncthreads();
#endif

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_COMBINE_RECV;

    // Clean up next buffer
    if (sm_id == 0 and warp_group_id == 0 and sub_warp_id == 0) {
        #pragma unroll
        for (int i = lane_id; i < num_next_clean_int; i += kWarpSize)
            next_clean[i] = 0;

        // Notify before executing `int_p`
        syncwarp();
        if (lane_id == 0)
            atomic_add_release_global(atomic_clean_flag, num_experts);
    }

    // Issue IBGDA sends
    if (responsible_expert_idx < num_experts) {
        const auto dst_rank = responsible_expert_idx / num_local_experts;
        const auto local_expert_idx = responsible_expert_idx % num_local_experts;
        const auto global_expert_idx = rank * num_local_experts + local_expert_idx;
        const auto layout = __ldg(layout_range + local_expert_idx * num_ranks + dst_rank);
        const auto local_x = reinterpret_cast<const int4*>(x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_bf16_int4;
        const auto local_src_info = src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
        const auto rdma_send_x_vec = reinterpret_cast<uint8_t*>(rdma_send_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_slot;

        // Unpack layout
        int offset, num_tokens_to_send;
        unpack2(layout, num_tokens_to_send, offset);

        // Issue IBGDA send
        for (int token_idx = offset + sub_warp_id; token_idx < offset + num_tokens_to_send; token_idx += kNumWarpsPerGroup) {
            const auto x_int4 = local_x + token_idx * hidden_bf16_int4;
            const auto rdma_send_type_row = reinterpret_cast<int*>(rdma_send_x_vec + token_idx * num_bytes_per_slot);
            const auto rdma_send_x_vec_row = reinterpret_cast<uint8_t*>(rdma_send_type_row + 4);

            // Copy directly to local rank, or copy to buffer and issue RDMA
            auto src_idx = __ldg(local_src_info + token_idx);
            const auto buf_ptr = reinterpret_cast<int64_t>(rdma_send_x_vec_row);
            const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) + (global_expert_idx * num_max_dispatch_tokens_per_rank + src_idx) * num_bytes_per_slot + sizeof(int4);
            if (dst_rank == rank) {
                const auto dst_int4_ptr = reinterpret_cast<int4*>(dst_ptr);
                UNROLLED_WARP_COPY(7, lane_id, hidden_bf16_int4, dst_int4_ptr, x_int4, ld_nc_global, st_na_global);
            } else {
                const auto buf_int4_ptr = reinterpret_cast<int4*>(buf_ptr);
                if (not zero_copy)
                    UNROLLED_WARP_COPY(7, lane_id, hidden_bf16_int4, buf_int4_ptr, x_int4, ld_nc_global, st_na_global);
                
                //nvshmemi_ibgda_put_nbi_warp(dst_ptr, buf_ptr, hidden * sizeof(gpu_bfloat16_t), dst_rank, local_expert_idx, lane_id, token_idx - offset);
#if defined(ROCM_DISABLE_CTX)
                    internode::shmemx_int8_put_nbi_warp(
#else
                    internode::shmem_ctx_schar_put_nbi_warp(ctx,
#endif
                    reinterpret_cast<signed char*>(dst_ptr), reinterpret_cast<signed char*>(buf_ptr), hidden * sizeof(gpu_bfloat16_t), dst_rank);
                if (num_ranks > 8){
#if defined(ROCM_DISABLE_CTX)
                    internode::shmem_fence();
#else
                    internode::shmem_ctx_quiet(ctx);
#endif
                }
            }
        }

        // Put finishing flag
        EP_STATIC_ASSERT(kNumWarpsPerGroup > 1, "Requires more than one warp per group");
#ifdef USE_ROCM
        if (lane_id == 0){
        volatile int ret = __hip_atomic_fetch_add(
            &sync_large_warp_counters[warp_group_id], 1,
            __ATOMIC_RELAXED, __HIP_MEMORY_SCOPE_WORKGROUP);
        }
        syncwarp();
        while (sync_large_warp_counters[warp_group_id] < (kNumWarpsPerGroup));
#else
        asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 1), "r"(kNumWarpsPerGroup * 32));
#endif
        if (sub_warp_id == 1 and lane_id == 0) {
            while (ld_acquire_global(atomic_clean_flag) == 0);
            if (dst_rank != rank) {
#ifdef USE_ROCM
#if defined(ROCM_DISABLE_CTX)
                internode::shmem_long_atomic_add(rdma_recv_flag + global_expert_idx, 1, dst_rank);
#else
                internode::shmem_ctx_long_atomic_add(ctx, rdma_recv_flag + global_expert_idx, 1, dst_rank);
#endif
#else
                nvshmemi_ibgda_amo_nonfetch_add(rdma_recv_flag + global_expert_idx, 1, dst_rank, local_expert_idx);
#endif
            } else {
                st_na_release(reinterpret_cast<int64_t*>(rdma_recv_flag + global_expert_idx), 1);
            }
            atomic_add_release_global(atomic_clean_flag, -1);
        }
        syncwarp();
    }

    // Receiving phase
    LOW_LATENCY_COMBINE_RECV:
    if ((phases & LOW_LATENCY_RECV_PHASE) == 0)
        return;

    // Wait all ranks to arrive and notify PCIe usage
    if (responsible_expert_idx < num_experts) {
        EP_STATIC_ASSERT(kNumWarpsPerGroup > 1, "Invalid number of warps per group");
        if (sub_warp_id == 0 and lane_id == 0){
            while (ld_acquire_global(reinterpret_cast<int*>(rdma_recv_flag + responsible_expert_idx)) == 0);
        }
    }
    grid_barrier(global_atomic_counter, num_sms);

    // Reduce tokens with FP8 cast
    EP_DEVICE_ASSERT(num_topk <= kWarpSize and hidden_bf16_int4 <= num_threads);
    EP_STATIC_ASSERT(kHidden % (kWarpSize * kNumElemsPerInt4) == 0, "Invalid vectorization");
    if (thread_id < hidden_bf16_int4) {
        for (int token_idx = sm_id; token_idx < num_combined_tokens; token_idx += num_sms) {
            // Read top-k indices and weights
            int reg_topk_idx[kNumMaxTopk];
            float reg_topk_weights[kNumMaxTopk];
            #pragma unroll
            for (int i = 0; i < num_topk; ++ i) {
                reg_topk_idx[i] = static_cast<int>(__ldg(topk_idx + token_idx * num_topk + i));
                reg_topk_weights[i] = __ldg(topk_weights + token_idx * num_topk + i);
            }

            float combined_values[kNumElemsPerInt4] = {0.0f};
            #pragma unroll
            for (int i = 0; i < num_topk; ++ i) if (reg_topk_idx[i] >= 0) {
                // Read from sources
                auto rdma_buffer_type = reinterpret_cast<const int*>(reinterpret_cast<uint8_t*>(rdma_recv_x) + (reg_topk_idx[i] * num_max_dispatch_tokens_per_rank + token_idx) * num_bytes_per_slot);
                auto rdma_buffer_row = reinterpret_cast<const uint8_t*>(rdma_buffer_type + 4);

                // Reduce
                auto x_vec = ld_nc_global(reinterpret_cast<const int4*>(rdma_buffer_row) + thread_id);
                const auto x_bf16 = reinterpret_cast<gpu_bfloat16_t*>(&x_vec);
                #pragma unroll
                for (int j = 0; j < kNumElemsPerInt4; ++ j)
                    combined_values[j] += static_cast<float>(x_bf16[j]) * reg_topk_weights[i];
            }

            // Write results
            int4& combined_int4 = *reinterpret_cast<int4*>(combined_values);
            auto combined_bf16 = reinterpret_cast<gpu_bfloat16_t*>(&combined_values);
            #pragma unroll
            for (int j = 0; j < kNumElemsPerInt4; ++ j)
                combined_bf16[j] = static_cast<gpu_bfloat16_t>(combined_values[j]);
            (reinterpret_cast<int4*>(combined_x) + token_idx * hidden_bf16_int4)[thread_id] = combined_int4;
        }
    }
#if !defined(ROCM_DISABLE_CTX)
    internode::shmem_wg_ctx_destroy(&ctx);
#endif
}

void combine(void* combined_x,
             void* rdma_recv_x, int64_t* rdma_recv_flag, void* rdma_send_x,
             const void* x, const int64_t* topk_idx, const float* topk_weights,
             const int* src_info, const int64_t* layout_range,
             int* global_atomic_counter,
             int64_t* next_clean, int num_next_clean_int,
             int num_combined_tokens, int hidden, int num_max_dispatch_tokens_per_rank,
             int num_topk, int num_experts, int rank, int num_ranks,
             void* workspace, hipStream_t stream,
             int phases, bool zero_copy) {
#ifdef USE_ROCM
    constexpr int kNumWarpsPerGroup = 4;
    constexpr int kNumWarpGroups = 4;
#else
    constexpr int kNumWarpsPerGroup = 10;
    constexpr int kNumWarpGroups = 3;
#endif
    constexpr int kNumMaxTopk = 9;

    const auto num_warps = kNumWarpGroups * kNumWarpsPerGroup;
    const auto num_sms = cell_div(num_experts, kNumWarpGroups);

    // Check workspace
    auto atomic_clean_flag = reinterpret_cast<int*>(workspace);
    EP_HOST_ASSERT(sizeof(int) <= NUM_WORKSPACE_BYTES);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopk);

#define COMBINE_LAUNCH_CASE(hidden) { \
auto combine_func = combine<kNumWarpGroups, kNumWarpsPerGroup, hidden, kNumMaxTopk>; \
LAUNCH_KERNEL_NON_COOPERATIVE(&cfg, combine_func, \
              combined_x, \
              rdma_recv_x, rdma_recv_flag, rdma_send_x, \
              x, topk_idx, topk_weights, src_info, layout_range, \
              global_atomic_counter, \
              next_clean, num_next_clean_int, \
              atomic_clean_flag, \
              num_combined_tokens, hidden, num_topk, \
              num_max_dispatch_tokens_per_rank, \
              num_experts, rank, num_ranks, \
              phases, zero_copy); } break

    SETUP_LAUNCH_CONFIG(num_sms, num_warps * kWarpSize, stream);
    SWITCH_HIDDEN(COMBINE_LAUNCH_CASE);
#undef COMBINE_LAUNCH_CASE
}

} // namespace internode_ll

} // namespace deep_ep